#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>
#include <iostream>
#include <vector>
//#include <float.h>


using namespace std;


#define N 512
#define M 512 //threads per block


__global__ void add(int *a, int *b, int *c, int n) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < n)
		c[index] = a[index] + b[index];
}


void sequential_ints(int* a, int size)
{
	for (int i = 0; i < size; i++)
		a[i] = i;
}


typedef vector<double*> Data;
typedef vector<double*> Centroids;
typedef vector<int> Output;

class kmeans {
public:
	kmeans(Data* data, int n, int dim) {
		this->data = data;
		this->n = n;
		this->dim = dim;
		this->k = 3;
		this->maxIter = 1000;
	}


	int* cluster(int k, int maxIter, Output out) {
		this->k = k;
		this->maxIter = maxIter;
		randCentroids();	//make this parallel
		int iter = 0;
		bool converged = false;
		while (!converged && iter < maxIter) {
			nearestCentroids();	//make this parallel
			converged = calcCentroids();	//make this parallel
			iter++;
		}

	}

	// fields
	Data* data;
	Centroids centroids;
	int n;
	int dim;
	int k;
	int maxIter;

private:
	double fRand(double min, double max) {
		double f = (double)rand() / RAND_MAX;
		return min + f * (max - min);
	}
	/* finds nearest centroids, must make parallel*/
	Output* nearestCentroids() {




	}

	/*recaclulates new centroids, must make parallel*/
	bool calcCentroids() {

	}
	/** I think we can paralellize this*/
	void randCentroids() {
		//get range of dataset
		vector<double> min = { DBL_MAX, DBL_MAX, DBL_MAX };
		vector<double> max = { -DBL_MAX, -DBL_MAX, -DBL_MAX };
		for (int i = 0; i < n; i++) {
			for (int j = 0; j < dim; j++) {
				if (data->at(i)[j] > max.at(j))
					max.at(j) = data->at(i)[j];
				if (data->at(i)[j] < min.at(j))
					min.at(j) = data->at(i)[j];
			}
		}

		double* tmp;
		for (int i = 0; i < k; i++) {
			tmp = new double[dim];
			for (int j = 0; j < dim; j++) {
				tmp[j] = fRand(min.at(k), max.at(k));
			}
		}
		centroids.push_back(tmp);
	}

};

int main(void) {
	int *a, *b, *c; // host copies of a, b, c
	int *d_a, *d_b, *d_c; // device copies of a, b, c
	int size = N * sizeof(int);
	// Alloc space for device copies of a, b, c
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);
	// Alloc space for host copies of a, b, c and setup input values
	a = (int *)malloc(size);
	sequential_ints(a, N);
	b = (int *)malloc(size);
	sequential_ints(b, N);
	c = (int *)malloc(size);
	// Copy inputs to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	// Launch add() kernel on GPU
	add << <(N + M - 1) / M, M >> > (d_a, d_b, d_c, N);
	// Copy result back to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
	// Cleanup
	for (int i = 0; i < N; i++)
		cout << a[i] << ", " << b[i] << ", " << c[i] << endl;
	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	return 0;
}

